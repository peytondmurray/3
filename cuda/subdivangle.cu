#include "hip/hip_runtime.h"
#include "stencil.h"
#include "subdivangle.h"

// Don't use this, something the preprocessor is doing makes it different than what I have currently.
// #define PI 3.14159265358979323846264338327950288419716939937510582097494459

extern "C" __global__ void subDivAngle(float* __restrict__ s, float* __restrict__ a, float* __restrict__ b,
                                       int shift, float dt, int Nx, int Ny, int Nz) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) return;

    if (ix + shift < Nx && ix + shift >= 0) {
        s[idx(ix, iy, iz)] = deltaAngle(a[idx(ix, iy, iz)] - b[idx(ix+shift, iy, iz)])/dt;
    } else {
        s[idx(ix, iy, iz)] = 0;
    }

    return;
}

extern "C" __device__ float deltaAngle(float dA) {
    float pi = 3.14159265358979323846264338327950288419716939937510582097494459;
    if (dA < -1*pi) {
        return 2*pi + dA;
    } else if (dA > pi) {
        return 2*pi - dA;
    }
    return dA;
}
